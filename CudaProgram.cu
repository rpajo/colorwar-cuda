#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "qdbmp.h"
#include "qdbmp.c"


__global__
void process (int *tabela, int*output, int* random, int vrsta, int pixlov, int offset) {
	int sosedi[3][4];
	int stBarv=-1;
	// x - index pixla -> Blok(vrsta) * število niti v bloku(širina) + zaporedna nit v vrsti
	int	x =  blockIdx.x*blockDim.x*3 + threadIdx.x*3;

	//sinhronizacija niti - verjentu nepotrebna
	__syncthreads();

	// Iskanje levega pixla
	if((x % (vrsta * 3)) != 0){
		stBarv++;
		sosedi[0][stBarv]= tabela[x-3];
		sosedi[1][stBarv]= tabela[x-2];
		sosedi[2][stBarv]= tabela[x-1];
	}

	// Iskanje zgornjega pixla
	if(x >= (vrsta * 3)){
		stBarv++;
		sosedi[0][stBarv]= tabela[x-(vrsta*3)];
		sosedi[1][stBarv]= tabela[x+1-(vrsta*3)];
		sosedi[2][stBarv]= tabela[x+2-(vrsta*3)];
	}

	// Iskanje desnega pixla
	//if( (x == 0) || (x % ((vrsta)*3) != (vrsta-1)*3)){
	if (((x + 3) % (vrsta * 3)) != 0) {
		stBarv++;
		sosedi[0][stBarv]= tabela[x+3];
		sosedi[1][stBarv]= tabela[x+4];
		sosedi[2][stBarv]= tabela[x+5];
	}

	// Iskanje spodnjega pixla
	//if(x < ((pixlov * 3) - (vrsta * 3))){
	if (x < ((vrsta - 1) * (vrsta * 3))) {
		stBarv++;
		sosedi[0][stBarv]= tabela[x+(vrsta*3)];
		sosedi[1][stBarv]= tabela[x+1+(vrsta*3)];
		sosedi[2][stBarv]= tabela[x+2+(vrsta*3)];
	}

	// če je index na intervalu slike v tabeli - zaradi Cude treba baje prevert da ne uzame krnek
	if(x < pixlov*3) {
		// random int iz tabele, vzamemo i-ti element (index pixla) + offset, da ni vedno isti random na pixlu
		int ran = random[blockIdx.x*blockDim.x + threadIdx.x + offset]%(stBarv+1);
		output[x] = sosedi[0][ran];
		output[x+1] = sosedi[1][ran];
		output[x+2] = sosedi[2][ran];
		//printf("%d\n", ran);

		// Izpis za debuggiranje
		/*printf("pixel: (%d, %d) r:%d sosedi: %d ->  [%d %d %d; %d %d %d; %d %d %d; %d %d %d] -> [%d %d %d]\n", blockIdx.x*blockDim.x, threadIdx.x, ran, stBarv+1,
			sosedi[0][0], sosedi[1][0], sosedi[2][0], sosedi[0][1], sosedi[1][1], sosedi[2][1],
			sosedi[0][2], sosedi[1][2], sosedi[2][2], sosedi[0][3], sosedi[1][3], sosedi[2][3],
			output[x], output[x+1], output[x+2]);
		*/
	}

}

int main(int argc, char* argv[]) {

	// Štoparica
	double diff = 0.0;
	time_t start;
    time_t stop;
    time(&start);


	BMP* bmp;
	BMP* nova;
	unsigned char r, g, b;
	int width, height;
	int x, y;

	printf("Vnesi stevilo iteraciji na GPU:\n");
	long cudaIteracije;
	scanf("%ld", &cudaIteracije);

	/* Preverimo, če je število vnešenih argumentov pravilno */
	if ( argc != 3 )
	{
		fprintf( stderr, "Uporaba: %s <vhodna slika> <izhodna slika>",
			argv[ 0 ] );
		return 0;
	}

	bmp = BMP_ReadFile( argv[ 1 ] );
	//BMP_CHECK_ERROR( stderr, -1 );

	width = BMP_GetWidth( bmp );
	height = BMP_GetHeight( bmp );

	srand ( time(NULL) );

	// alociranje pomnilnika
	int *tabela1D;
	int *rezultat;
	int *random;
	int *cudaRandom;
	int *cudaInput;
	int *cudaOutput;

	tabela1D = (int*)malloc(width*height*3*sizeof(int));
	rezultat = (int*)malloc(width*height*3*sizeof(int));
	random = (int*)malloc(width*height*3*sizeof(int));
	hipMalloc(&cudaRandom, (cudaIteracije+width*height)*sizeof(int));
	hipMalloc(&cudaOutput, width*height*3*sizeof(int));
	hipMalloc(&cudaInput, width*height*3*sizeof(int));

	//preberi RGB vrednosti vsakega pixla na sliki v 1D tabelo
	int j=0;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width; x++) {
			BMP_GetPixelRGB( bmp, x, y, &r, &g, &b );
			/*printf("%d) %u %u %u\n", j, r, g, b);
			j+=3;*/
			tabela1D[y*width*3+x*3] = (int)r;
			tabela1D[y*width*3+x*3+1] = (int)g;
			tabela1D[y*width*3+x*3+2] = (int)b;
		}
	}

	// Generiranje random intov v tabelo, ki jo poščjemo na gpu
	for(j = 0; j < height*width+cudaIteracije; j++) {
		random[j] = rand();
	}

	//prenos podatkov iz Hosta na GPU
	hipMemcpy(cudaInput, tabela1D, width*height*3*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaRandom, random, (cudaIteracije + width*height)*sizeof(int), hipMemcpyHostToDevice);


	long i = 0;
	char name[64];
	char datoteka[64];
	int counter = 1;

	// Klicanje glavne metode v zanki
	for(i = 0; i < cudaIteracije; i++) {
		//printf("iteracija: %d\n", i+1);
		process<<<height, width>>>(cudaInput, cudaOutput, cudaRandom, width, width*height, i);

		// Prenesemo generirano sliko iz GPU na Host in jo nazaj pošjemo kot izvirno sliko
		hipMemcpy(rezultat, cudaOutput, width*height*3*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(cudaInput, rezultat, width*height*3*sizeof(int), hipMemcpyHostToDevice);
		/*j = 0;
		for(x = 0; x < height*width*3; x++) {
			printf("%d ", rezultat[x]);
			j++;
			if(j == 3) {
				j = 0;
				printf("\n");
			}
		}
		printf("------------------------\n");*/

		// Shranimo vsako 5 sliko v mapo Izhodi
		if(i != 0 && i%5 == 0 ) {
			nova = BMP_Create(width, height, 24);
			for(y = 0; y < height; y++) {
				for(x = 0; x < width; x++) {
					BMP_SetPixelRGB(nova, x, y, (unsigned char)rezultat[y*width*3+x*3],
												(unsigned char)rezultat[y*width*3+x*3+1],
												(unsigned char)rezultat[y*width*3+x*3+2]);
				}
			}

			strcpy(name, "Vojne/izhodi/");
			sprintf(datoteka, "%d", i);
			strcat(name, datoteka);
			strcat(name, ".bmp");

			BMP_WriteFile( nova, name);
			}
	}

	//Shranjevanje zadnje slike - nepotrebno, če shranjujemo sproti
	/*hipMemcpy(rezultat, cudaOutput, width*height*3*sizeof(int), hipMemcpyDeviceToHost);


	nova = BMP_Create(width, height, 24);


	for(y = 0; y < height; y++) {
		for(x = 0; x < width; x++) {
			BMP_SetPixelRGB(nova, x, y, (unsigned char)rezultat[y*width*3+x*3],
										(unsigned char)rezultat[y*width*3+x*3+1],
										(unsigned char)rezultat[y*width*3+x*3+2]);
		}
	}

	BMP_WriteFile(nova, argv[2]);
	BMP_CHECK_ERROR(stdout, -2);*/

	// Sprostimo pomnilnik
	free(tabela1D);
	free(random);
	hipFree(cudaRandom);
	hipFree(cudaInput);
	hipFree(cudaOutput);

	// Ustavimo štoparico
	time(&stop);
  	diff = difftime(stop, start);
  	printf("Runtime: %g\n", diff);

	return 0;
}