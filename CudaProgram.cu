#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
//#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "qdbmp.h"
#include "qdbmp.c"


__global__
void process (int *tabela, int*output, int* random, int vrsta, int pixlov) {
	//__shared__ int block[2*vrsta*3];
	int sosedi[3][4];
	int stBarv=-1;
	int	x =  blockIdx.x*blockDim.x + threadIdx.x;

	if((x-3)>=0){
		/* Preberemo RGB vrednosti x-1, y pike */
		stBarv++;
		sosedi[stBarv][0]= tabela[x-3];
		sosedi[stBarv][1]= tabela[x-2];
		sosedi[stBarv][2]= tabela[x-1];
	}
				
	if((x-(vrsta*3))>=0){
		stBarv++;
		sosedi[stBarv][0]= tabela[x-(vrsta*3)];
		sosedi[stBarv][1]= tabela[x+1-(vrsta*3)];
		sosedi[stBarv][2]= tabela[x+2-(vrsta*3)];
	}

	if((x+3) < pixlov){
		stBarv++;
		sosedi[stBarv][0]= tabela[x+3];
		sosedi[stBarv][1]= tabela[x+4];
		sosedi[stBarv][2]= tabela[x+5];
	}

	if((x+(vrsta*3)) < pixlov){
		stBarv++;
		sosedi[stBarv][0]= tabela[x+(vrsta*3)];
		sosedi[stBarv][1]= tabela[x+1+(vrsta*3)];
		sosedi[stBarv][2]= tabela[x+2+(vrsta*3)];
	}
	
	
	int ran = random[x];
	output[x] = sosedi[ran][0];
	output[x+1] = sosedi[ran][1];
	output[x+2] = sosedi[ran][2];


	}

int main(int argc, char* argv[]) {

	double diff = 0.0;
	time_t start;
    time_t stop;
    time(&start);


	BMP* bmp;
	BMP* nova;
	unsigned char r, g, b; 
	int width, height; 
	int x, y; 

	/* Preverimo, če je število vnešenih argumentov pravilno */
	if ( argc != 3 )
	{
		fprintf( stderr, "Uporaba: %s <vhodna slika> <izhodna slika>",
			argv[ 0 ] );
		return 0;
	}

	bmp = BMP_ReadFile( argv[ 1 ] );
	//BMP_CHECK_ERROR( stderr, -1 );
	
	width = BMP_GetWidth( bmp );
	height = BMP_GetHeight( bmp );

	srand ( time(NULL) );

	// alociranje pomnilnika
	int *tabela1D;
	int *rezultat;
	int *random;
	int *cudaRandom;
	int *cudaInput;
	int *cudaOutput;

	tabela1D = (int*)malloc(width*height*3*sizeof(int));
	rezultat = (int*)malloc(width*height*3*sizeof(int));
	random = (int*)malloc(width*height*3*sizeof(int));
	hipMalloc(&cudaRandom, width*height*sizeof(int));
	hipMalloc(&cudaOutput, width*height*3*sizeof(int));
	hipMalloc(&cudaInput, width*height*3*sizeof(int));

	//preberi RGB vrednosti vsakega pixla na sliki v 1D tabelo
	for(y = 0; y < height; y++) {
		for(x = 0; x < width; x++) {
			BMP_GetPixelRGB( bmp, x, y, &r, &g, &b );
			// printf("%u %u %u\n", r, g, b);
			// printf("-> %d %d %d\n", (int)r, (int)g, (int)b);
			tabela1D[y*width+x*3] = (int)r;
			tabela1D[y*width+x*3+1] = (int)g;
			tabela1D[y*width+x*3+2] = (int)b;
			random[y*width+x] = rand()%4;
			//printf("%d, %d, %d\n", y*width+x*3, y*width+x*3+1, y*width+x*3+2);
		}
	}

	hipMemcpy(cudaInput, tabela1D, width*height*3*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaRandom, random, width*height*sizeof(int), hipMemcpyHostToDevice);




	process<<<height , width*3>>>(cudaInput, cudaOutput, cudaRandom, width, width*height);

	hipMemcpy(rezultat, cudaOutput, width*height*3*sizeof(int), hipMemcpyDeviceToHost);


	nova = BMP_Create(width, height, 24);

	for(y = 0; y < height; y++) {
		for(x = 0; x < width; x++) {
			BMP_SetPixelRGB(nova, x, y, (unsigned char)rezultat[y*width+x*3], 
										(unsigned char)rezultat[y*width+x*3+1], 
										(unsigned char)rezultat[y*width+x*3+2]);
		}
	}

	BMP_WriteFile(nova, argv[2]);
	BMP_CHECK_ERROR(stdout, -2);

	free(tabela1D);
	free(random);
	hipFree(cudaRandom);
	hipFree(cudaInput);
	hipFree(cudaOutput);

	time(&stop);
  	diff = difftime(stop, start);
  	printf("Runtime: %g\n", diff);

	return 0;
}