#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "qdbmp.h"
#include "qdbmp.c"


__global__
void process (int *tabela, int* random, int vrsta, int pixlov, int randOffset) {
	int sosedi[3][4];	
	int stBarv=-1;
	long zamik = pixlov*3 * randOffset;
	// x - index pixla -> Blok(vrsta) * število niti v bloku(širina) + zaporedna nit v vrsti
	long	x =  blockIdx.x*blockDim.x*3 + threadIdx.x*3 + pixlov*3 * (randOffset-1);
	long y = blockIdx.x*blockDim.x*3 + threadIdx.x*3;
	//printf("blockIdx.x: %d; blockDim.x: %d; threadIdx.x: %d -> x: %d\n", blockIdx.x, blockDim.x, threadIdx.x, x);

	// Iskanje levega pixla
	if(y % (vrsta*3) != 0){
		stBarv++;
		sosedi[0][stBarv]= tabela[x-3];
		sosedi[1][stBarv]= tabela[x-2];
		sosedi[2][stBarv]= tabela[x-1];
	}
	//else printf("Levi rob %d %% %d == 0\n", y, vrsta*3);
		
	// Iskanje zgornjega pixla		
	if(y >= vrsta*3){
		stBarv++;
		sosedi[0][stBarv]= tabela[x-(vrsta*3)];
		sosedi[1][stBarv]= tabela[x+1-(vrsta*3)];
		sosedi[2][stBarv]= tabela[x+2-(vrsta*3)];
	}
	//else printf("Zgornji rob x:%d\n", y);

	// Iskanje desnega pixla
	if( (y == 0) || (y % ((vrsta)*3) != (vrsta-1)*3)){
		stBarv++;
		sosedi[0][stBarv]= tabela[x+3];
		sosedi[1][stBarv]= tabela[x+4];
		sosedi[2][stBarv]= tabela[x+5];
	}
	//else printf("Desni rob x:%d\n", y);

	// Iskanje spodnjega pixla
	if(y < pixlov*3 - vrsta*3){
		stBarv++;
		sosedi[0][stBarv]= tabela[x+(vrsta*3)];
		sosedi[1][stBarv]= tabela[x+1+(vrsta*3)];
		sosedi[2][stBarv]= tabela[x+2+(vrsta*3)];
	}
	//else printf("Spodnji rob x: %d\n", y);

	// če je index na intervalu slike v tabeli - zaradi Cude treba baje prevert da ne uzame krnek
	//if(x < pixlov*3*(randOffset+1)) {
		// random int iz tabele, vzamemo i-ti element (index pixla) + offset, da ni vedno isti random na pixlu
		int ran = random[blockIdx.x*blockDim.x + threadIdx.x + (randOffset-1)]%(stBarv+1);
		tabela[pixlov*3 + x] = sosedi[0][ran];
		tabela[pixlov*3 + x+1] = sosedi[1][ran];
		tabela[pixlov*3 + x+2] = sosedi[2][ran];
		//printf("%d\n", ran);

		// Izpis za debuggiranje
		/*if (true) {
			printf("[%d] pixel: (%d, %d) r:%d sosedi: %d ->  [%d %d %d; %d %d %d; %d %d %d; %d %d %d] -> [%d %d %d] --> Zamik: %d Zapis v: %d,%d,%d\n", randOffset, blockIdx.x, threadIdx.x, ran, stBarv+1,
				sosedi[0][0], sosedi[1][0], sosedi[2][0], sosedi[0][1], sosedi[1][1], sosedi[2][1], 
				sosedi[0][2], sosedi[1][2], sosedi[2][2], sosedi[0][3], sosedi[1][3], sosedi[2][3],
				tabela[zamik + x], tabela[zamik + x + 1], tabela[zamik + x + 2], zamik, zamik + x,  zamik + x+1, zamik + x+2);
		}
		*/		
	//}

}

int main(int argc, char* argv[]) {

	// Štoparica
	double diff = 0.0;
	time_t start;
    time_t stop;
    time(&start);


	BMP* bmp;
	BMP* nova;
	unsigned char r, g, b; 
	int width, height; 
	int x, y; 

	printf("Vnesi stevilo iteraciji na GPU:\n");
	long cudaIteracije;
	scanf("%ld", &cudaIteracije);

	/* Preverimo, če je število vnešenih argumentov pravilno */
	if ( argc != 3 )
	{
		fprintf( stderr, "Uporaba: %s <vhodna slika> <izhodna slika>",
			argv[ 0 ] );
		return 0;
	}

	bmp = BMP_ReadFile( argv[ 1 ] );
	//BMP_CHECK_ERROR( stderr, -1 );
	
	width = BMP_GetWidth( bmp );
	height = BMP_GetHeight( bmp );

	srand ( time(NULL) );

	// alociranje pomnilnika
	int *tabela1D;
	int *rezultat;
	int *random;
	int *cudaRandom;
	int *cudaTabela;

	tabela1D = (int*)malloc(width*height*3*sizeof(int));
	rezultat = (int*)malloc((cudaIteracije+1) * width*height*3*sizeof(int));
	random = (int*)malloc(width*height*3*sizeof(int));
	hipMalloc(&cudaRandom, (cudaIteracije+width*height)*sizeof(int));
	hipMalloc(&cudaTabela, (cudaIteracije+1) * width*height*3*sizeof(int));

	//preberi RGB vrednosti vsakega pixla na sliki v 1D tabelo
	int j=0;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width; x++) {
			BMP_GetPixelRGB( bmp, x, y, &r, &g, &b );
			/*printf("%d) %u %u %u\n", j, r, g, b);
			j+=3;*/
			tabela1D[y*width*3+x*3] = (int)r;
			tabela1D[y*width*3+x*3+1] = (int)g;
			tabela1D[y*width*3+x*3+2] = (int)b;
		}
	}

	// Generiranje random intov v tabelo, ki jo poščjemo na gpu
	for(j = 0; j < height*width+cudaIteracije; j++) {
		random[j] = rand();
	}

	//prenos podatkov iz Hosta na GPU
	hipMemcpy(cudaTabela, tabela1D, width*height*3*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cudaRandom, random, (cudaIteracije + width*height)*sizeof(int), hipMemcpyHostToDevice);


	long i = 0;
	char name[64];
	char datoteka[64];
	int counter = 1;
	int rdeca = 0;
  	int modra = 0;
  	int zelena = 0;

	// Klicanje glavne metode v zanki
	for(i = 1; i <= cudaIteracije; i++) {
		//printf("iteracija: %d\n", i+1);
		process<<<height, width>>>(cudaTabela, cudaRandom, width, width*height, i);

	}

	hipMemcpy(rezultat, cudaTabela, (cudaIteracije+1)* width*height*3*sizeof(int), hipMemcpyDeviceToHost);


	nova = BMP_Create(width, height, 24);

	for(i = 0; i < cudaIteracije+1; i++) {
		int offset = i*width*height*3;
		if(i%200 == 0) {
			for(y = 0; y < height; y++) {
				for(x = 0; x < width; x++) {
					BMP_SetPixelRGB(nova, x, y, (unsigned char)rezultat[offset + y*width*3+x*3], 
												(unsigned char)rezultat[offset + y*width*3+x*3+1], 
												(unsigned char)rezultat[offset + y*width*3+x*3+2]);
				/*printf("[%d, %d, %d]\n", (unsigned char)rezultat[offset + y*width*3+x*3], 
											(unsigned char)rezultat[offset + y*width*3+x*3+1], 
											(unsigned char)rezultat[offset + y*width*3+x*3+2]);*/
				}
			}
			strcpy(name, "Vojne/izhodi/");
			sprintf(datoteka, "%d", i);
			strcat(name, datoteka);
			strcat(name, ".bmp");
			BMP_WriteFile( nova, name);
			BMP_CHECK_ERROR(stdout, -2);
		}
	}

	

	// Sprostimo pomnilnik
	free(tabela1D);
	free(random);
	hipFree(cudaRandom);
	hipFree(cudaTabela);
	BMP_Free(nova);

	// Ustavimo štoparico
	time(&stop);
  	diff = difftime(stop, start);
  	printf("St. iteracij: %ld Runtime: %g\n", i, diff);

	return 0;
}